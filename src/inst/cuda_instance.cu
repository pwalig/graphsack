#include "cuda_instance.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_instance_memory.cuh"

__constant__ uint32_t gs::cuda::inst::limits[GS_CUDA_INST_MAXM];
__constant__ uint32_t gs::cuda::inst::values[GS_CUDA_INST_MAXN];
__constant__ uint32_t gs::cuda::inst::weights[GS_CUDA_INST_MAXN * GS_CUDA_INST_MAXM];
__constant__ uint64_t gs::cuda::inst::adjacency[GS_CUDA_INST_MAXN];

void gs::cuda::inst::copy_to_symbol(const instance64<uint32_t, uint32_t>& inst)
{
	hipMemcpyToSymbol(HIP_SYMBOL(limits), inst.limits_data(), inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(values), inst.values_data(), inst.size() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(weights), inst.weights_data(), inst.size() * inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(adjacency), inst.graph_data(), inst.size() * sizeof(uint64_t));
	hipDeviceSynchronize();
}

__device__  bool gs::cuda::inst::has_connection_to(const uint64_t* adjacency, uint32_t from, uint32_t to) {
	if (adjacency[from] & (uint64_t(1) << to)) return true;
	else return false;
}

