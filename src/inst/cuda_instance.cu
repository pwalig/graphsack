#include "cuda_instance.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_instance_memory.cuh"

void gs::inst::cuda::copy_to_symbol(const instance<uint32_t, uint32_t>& inst)
{
	hipMemcpyToSymbol(HIP_SYMBOL(limits), inst.limits_data(), inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(values), inst.values_data(), inst.size() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(weights), inst.weights_data(), inst.size() * inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(adjacency), inst.graph_data(), inst.size() * sizeof(uint64_t));
	hipDeviceSynchronize();
}
