#include "cuda_instance.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_instance_memory.cuh"

void gs::cuda::inst::copy_to_symbol(const instance64<uint32_t, uint32_t>& inst)
{
	hipMemcpyToSymbol(HIP_SYMBOL(limits), inst.limits_data(), inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(values), inst.values_data(), inst.size() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(weights), inst.weights_data(), inst.size() * inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(adjacency), inst.graph_data(), inst.size() * sizeof(uint64_t));
	hipDeviceSynchronize();
}

__device__  bool gs::cuda::inst::has_connection_to(const uint64_t* adjacency, uint32_t from, uint32_t to) {
	if (adjacency[from] & (uint64_t(1) << to)) return true;
	else return false;
}

