#include "cuda_instance.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "cuda_instance.cuh"

__constant__ uint32_t gs::cuda::inst::limits[GS_CUDA_INST_MAXM];
__constant__ uint32_t gs::cuda::inst::values[GS_CUDA_INST_MAXN];
__constant__ uint32_t gs::cuda::inst::weights[GS_CUDA_INST_MAXN * GS_CUDA_INST_MAXM];
__constant__ uint64_t gs::cuda::inst::adjacency[GS_CUDA_INST_MAXN];

void gs::cuda::inst::copy_to_symbol(const instance64<uint32_t, uint32_t>& inst)
{
	hipMemcpyToSymbol(HIP_SYMBOL(limits), inst.limits().data(), inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(values), inst.values().data(), inst.size() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(weights), inst.weights().data(), inst.size() * inst.dim() * sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(adjacency), inst.graph_data(), inst.size() * sizeof(uint64_t));
	hipDeviceSynchronize();
}

