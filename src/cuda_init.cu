#include "cuda_init.h"

#include "hip/hip_runtime.h"

void gs::cuda::init() {
    int nDevices;
	int bestDevice = -1;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//printf("Device Number: %d\n", i);
		//printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		//printf("  Threads Per Block: %d\n", prop.maxThreadsPerBlock);
		//printf("  Can Map Host Memory: %d\n", prop.canMapHostMemory);
		//printf("  totalConstMem: %llu\n", prop.totalConstMem);
		//printf("  totalGlobalMem: %llu\n" ,prop.totalGlobalMem);	
	}

	hipSetDevice(0);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
}