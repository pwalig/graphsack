#include "cuda_init.h"

#include "hip/hip_runtime.h"
#include <stdexcept>

gs::cuda::device_properties gs::cuda::init() {
    int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		device_properties current = {
			prop.totalGlobalMem,
			prop.sharedMemPerBlock,
			prop.totalConstMem,
			prop.maxThreadsPerBlock,
			prop.canMapHostMemory
		};
		hipSetDevice(i);
		return current;
	}

	throw std::runtime_error("No devices found");
}