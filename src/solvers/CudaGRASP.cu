#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>

#include <stdexcept>
#include <cmath>
#include <vector>

#include "CudaGRASP.hpp"
#include "cuda_reductions.cuh"
#include "cuda_greedy_utils.cuh"
#include "../cuda/buffer.cuh"
#include "../cuda/error_wrapper.cuh"
#include "../cuda/curand_wrapper.cuh"
#include "../inst/cuda_instance.cuh"
#include "../res/cuda_solution.cuh"
#include "../cuda_structure_check.cuh"


namespace gs {
	namespace cuda {
		namespace solver {
			const std::string GRASP32::name = "CudaGRASP32";
			const std::string GRASP64::name = "CudaGRASP64";

			namespace grasp {
				template <typename result_type, typename index_type>
				__global__ void cycle_kernel(
					index_type N, uint32_t M, hiprandStateMtgp32* random_state, 
					uint32_t* value_memory, uint32_t* weight_memory, result_type* result_memory, index_type* stack_memory,
					index_type* sorted, index_type choose_from
				) {
					const size_t id = blockIdx.x * blockDim.x + threadIdx.x;

					// setup
					value_memory[id] = 0;
					result_memory[id] = 0;
					for (uint32_t wid = 0; wid < M; ++wid) weight_memory[M * id + wid] = 0;

					// construct solution
					for (index_type left = N; left > 0; --left) {
						index_type left_index = ::hiprand(&random_state[blockIdx.x]) % (left < choose_from ? left : choose_from);

						index_type sorted_index = 0;
						while (res::has(result_memory[id], sorted[sorted_index])) ++sorted_index;

						for (index_type i = 0; i < left_index; ++i) {
							++sorted_index;
							while (res::has(result_memory[id], sorted[sorted_index])) ++sorted_index;
						}

						index_type to_add = sorted[sorted_index];
						bool fitting = true;
						for (uint32_t wid = 0; wid < M; ++wid) {
							if (weight_memory[M * id + wid] + inst::weights<uint32_t>()[M * to_add + wid] > inst::limits<uint32_t>()[wid]) {
								fitting = false;
								break;
							}
						}
						if (fitting) {
							res::add(result_memory[id], to_add);
							if (!is_cycle_possible_recursive<result_type, uint32_t, index_type>(
								result_memory[id], N, M
							)) res::remove(result_memory[id], to_add);
							else {
								for (uint32_t wid = 0; wid < M; ++wid)
									weight_memory[M * id + wid] += inst::weights<uint32_t>()[M * to_add + wid];
								value_memory[id] += inst::values<uint32_t>()[to_add];
							}
						}
					}

					// do reduction
					GS_CUDA_REDUCTIONS_PICK(result_type, id, value_memory, result_memory)
				}

				template <typename result_type>
				res::solution<result_type> runner(
					const inst::instance<result_type, uint32_t, uint32_t>& instance, uint32_t blocksCount
				) {
					using index_type = typename inst::instance<result_type, uint32_t, uint32_t>::index_type;

					if (blocksCount > 200) throw std::invalid_argument("cudaGRASP blocksCount limit of 200 exeeded");
					uint32_t threadsPerBlock = 256;
					size_t totalThreads = threadsPerBlock * blocksCount;

					inst::copy_to_symbol(instance);

					buffer<uint32_t> weight_value(totalThreads * (instance.dim() + 1));
					buffer<index_type> index_memory(totalThreads * instance.size() + instance.size());
					buffer<result_type> result_memory(totalThreads);

					buffer<hiprandStateMtgp32> random_states(blocksCount);
					buffer<mtgp32_kernel_params> kernel_params(1);

					hiprand::MakeMTGP32Constants(kernel_params);
					hiprand::MakeMTGP32KernelState(random_states, kernel_params, blocksCount, time(NULL));

					except::DeviceSynchronize();
					//sort::in_order<index_type><<<1, 64>>>(index_memory.data(), instance.size());
					sort::reverse_order<index_type><<<1, 64>>>(index_memory.data(), instance.size());
					//sort::by_value<index_type, uint32_t><<<1, 64>>>(index_memory.data(), values, instance.size());
					except::DeviceSynchronize();
					//index_memory.debug_print(0, instance.size(), 1);

					cycle_kernel<result_type, index_type><<<blocksCount, threadsPerBlock>>>(
						instance.size(), instance.dim(), random_states.data(),
						weight_value.data(), weight_value.data() + totalThreads,
						result_memory.data(), index_memory.data() + instance.size(),
						index_memory.data(), instance.size() / 2
					);
					except::DeviceSynchronize();

					if (blocksCount > 1) {
						blocksCount /= 2;
						reductions::pick<result_type, uint32_t><<<1, blocksCount>>> (
							weight_value.data(),
							result_memory.data(),
							threadsPerBlock,
							totalThreads
						);
						except::DeviceSynchronize();
					}

					res::solution<result_type> result(instance.size());
					result_memory.get(&result._data);

					return result;
				}

				res::solution32 runner32(
					const inst::instance32<uint32_t, uint32_t>& instance, uint32_t blocksCount
				) {
					return runner<uint32_t>(instance, blocksCount);
				}

				res::solution64 runner64(
					const inst::instance64<uint32_t, uint32_t>& instance, uint32_t blocksCount
				) {
					return runner<uint64_t>(instance, blocksCount);
				}
			}
		}
	}
}

