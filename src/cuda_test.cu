#include "hip/hip_runtime.h"
#include "cuda_test.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void gs::cuda::test()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

void gs::cuda::info::print_json()
{
    hipDeviceProp_t  prop;
	int count;
	printf("{\n");
	hipGetDeviceCount(&count);
	// printf("\"DeviceCount\":\"%d\",\n",count);
	printf("\t\"Devices\":[");
	for (int i=0;i<count;i++) {
		printf("\n\t{\n");
		hipGetDeviceProperties(&prop,i);
		printf("\t\t\"name\":\"%s\",\n",prop.name);
		printf("\t\t\"major\":\"%d\",\n",prop.major);
		printf("\t\t\"minor\":\"%d\",\n",prop.minor);
		printf("\t\t\"computeMode\":\"%d\",\n",prop.computeMode);
		printf("\t\t\"integrated\":\"%d\",\n",prop.integrated);
		printf("\t\t\"tccDriver\":\"%d\",\n",prop.tccDriver);
		printf("\t\t\"ECCEnabled\":\"%d\",\n",prop.ECCEnabled);
		printf("\t\t\"deviceOverlap\":\"%d\",\n",prop.deviceOverlap); // Przeplatanie urz�dze�
		printf("\t\t\"concurrentKernels\":\"%d\",\n",prop.concurrentKernels);
		printf("\t\t\"kernelExecTimeoutEnabled\":\"%d\",\n",prop.kernelExecTimeoutEnabled);
		printf("\t\t\"canMapHostMemory\":\"%d\",\n",prop.canMapHostMemory);
		printf("\t\t\"multiProcessorCount\":\"%d\",\n",prop.multiProcessorCount);
		printf("\t\t\"warpSize\":\"%d\",\n",prop.warpSize);
		printf("\t\t\"clockRate\":\"%d\",\n",prop.clockRate);
		printf("\t\t\"maxThreadsPerBlock\":\"%d\",\n",prop.maxThreadsPerBlock);
		printf("\t\t\"maxThreadsDim\":[%d,%d,%d],\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("\t\t\"maxGridSize\":[%d,%d,%d],\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\t\t\"maxTexture1D\":\"%d\",\n",prop.maxTexture1D);
		printf("\t\t\"maxTexture2D\":[%d,%d],\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);
		printf("\t\t\"maxTexture3D\":[%d,%d,%d],\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
		printf("\t\t\"pciBusID\":\"%d\",\n",prop.pciBusID);
		printf("\t\t\"pciDeviceID\":\"%d\",\n",prop.pciDeviceID);
		printf("\t\t\"regsPerBlock\":\"%d\",\n",prop.regsPerBlock);
		// size_t
		printf("\t\t\"memPitch\":\"%llu\",\n",prop.memPitch);
		printf("\t\t\"surfaceAlignment\":\"%llu\",\n",prop.surfaceAlignment);
		printf("\t\t\"textureAlignment\":\"%llu\",\n",prop.textureAlignment);
		printf("\t\t\"totalConstMem\":\"%llu\",\n",prop.totalConstMem);
		printf("\t\t\"totalGlobalMem\":\"%llu\",\n",prop.totalGlobalMem);	
		printf("\t\t\"sharedMemPerBlock\":\"%llu\",\n",prop.sharedMemPerBlock);
		// size_t
		printf("\t\t\"memPitch\":\"%zu\",\n",prop.memPitch);
		printf("\t\t\"surfaceAlignment\":\"%zu\",\n",prop.surfaceAlignment);
		printf("\t\t\"textureAlignment\":\"%zu\",\n",prop.textureAlignment);
		printf("\t\t\"totalConstMem\":\"%zu\",\n",prop.totalConstMem);
		printf("\t\t\"totalGlobalMem\":\"%zu\",\n",prop.totalGlobalMem);
		printf("\t\t\"sharedMemPerBlock\":\"%zu\",\n",prop.sharedMemPerBlock);
		printf("\t\t\"canMapHostMemory\":\"%d\",\n",prop.canMapHostMemory);
		printf("\t\t\"l2CacheSize\":\"%d\"\n",prop.l2CacheSize);
		printf("\t\t\"persistingL2CacheMaxSize\":\"%d\"\n",prop.persistingL2CacheMaxSize);
		printf("\t\t\"regsPerBlock\":\"%d\"\n",prop.regsPerBlock);
		if (i<count-1) printf("\t},\n"); else printf("\t}\n");
	}
	printf("\t]\n}\n");
}

void gs::cuda::info::print()
{
	printf("Cuda Version: %d\n", CUDART_VERSION);

    int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("  Threads Per Block: %d\n", prop.maxThreadsPerBlock);
		printf("  Can Map Host Memory: %d\n", prop.canMapHostMemory);
		printf("  totalConstMem: %llu\n", prop.totalConstMem);
		printf("  totalGlobalMem: %llu\n" ,prop.totalGlobalMem);	
	}
}
